#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

#define N 1000000

__global__ void add(int* A, int* B, int* C) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    int* A, * B, * C;
    hipMallocManaged(&A, N * sizeof(int));
    hipMallocManaged(&B, N * sizeof(int));
    hipMallocManaged(&C, N * sizeof(int));

    for (int i = 0; i < N; i++) A[i] = rand() % 100, B[i] = rand() % 100;

    add <<<(N + 255) / 256, 256 >>> (A, B, C);
    hipDeviceSynchronize();

    for (int i = 0; i < 5; i++)
        cout << A[i] << "+" << B[i] << "=" << C[i] << endl;

    hipFree(A); hipFree(B); hipFree(C);
    return 0;
}





#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

#define N 5  // small size for easy printing

__global__ void mul(int* A, int* B, int* C) {
    int r = threadIdx.y + blockIdx.y * blockDim.y;
    int c = threadIdx.x + blockIdx.x * blockDim.x;
    if (r < N && c < N) {
        int sum = 0;
        for (int k = 0; k < N; k++)
            sum += A[r * N + k] * B[k * N + c];
        C[r * N + c] = sum;
    }
}

void printMatrix(int* M, string name) {
    cout << name << ":\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            cout << M[i * N + j] << " ";
        cout << endl;
    }
    cout << endl;
}

int main() {
    int* A, * B, * C;
    hipMallocManaged(&A, N * N * sizeof(int));
    hipMallocManaged(&B, N * N * sizeof(int));
    hipMallocManaged(&C, N * N * sizeof(int));b;/

    for (int i = 0; i < N * N; i++) A[i] = rand() % 10, B[i] = rand() % 10;

    dim3 threads(N, N);
    mul << <1, threads >> > (A, B, C);
    hipDeviceSynchronize();

    printMatrix(A, "Matrix A");
    printMatrix(B, "Matrix B");
    printMatrix(C, "Matrix C (Result)");

    hipFree(A); hipFree(B); hipFree(C);
    return 0;
}